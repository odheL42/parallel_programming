#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <cstring>
#include <cmath>
#include <ctime>
#include <nvtx3/nvToolsExt.h>

//следующие фнкции будут вызываться на host, но выполняться на device, поэтому используем __global__
__global__ void getErrorMatrix(double* A, double* Anew, double* end, size_t size){
    //вычисление ошибки
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > size * size) return;
	
	end[idx] = std::abs(Anew[idx] - A[idx]);
}
__global__ void calculateMatrix(double* A, double* Anew, size_t size){
    //Основные вычисления
	unsigned int j = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y;
	if (i * size + j > (size) * (size)) return;
///!!!!!!!!!!!!!!!!!
	if(!((j == 0 || i == 0 || j == size - 1 || i == size - 1)))
	{
		Anew[i * size + j] = 0.25 * (A[i * size + j - 1] + A[(i - 1) * size + j] +
							A[(i + 1) * size + j] + A[i * size + j + 1]);		
	}
}
//проверить выход за границы памяти ++++++++++++=


int main(int argc, char** argv) {
    int  iters = 1000000, grid_size = 128;
    double accuracy = 1e-6;
        for (int i = 0; i < argc - 1; i++) {
        std::string arg = argv[i];
        if (arg == "-accuracy") {
            std::string dump = std::string(argv[i + 1]);
            accuracy = std::stod(dump);
        }
        else if (arg == "-grid") {
            grid_size = std::stoi(argv[i + 1]);
        }
        else if (arg == "-iters") {
            iters = std::stoi(argv[i + 1]);
        }}
    
    int full_size = grid_size * grid_size;
    double step = 1.0 * (20 - 10) / (grid_size - 1);
    //auto* A = new double[full_size];
    //auto* Anew = new double[full_size];
    //std::memset(A, 0, sizeof(double) * full_size);

	double* A;
	double* Anew;

	hipHostMalloc(&A, full_size * sizeof(double));
	hipHostMalloc(&Anew, full_size * sizeof(double));
	
	std::memset(A, 0, full_size * sizeof(double));

    //инициализируем углы
    A[0] = 10;
    A[grid_size - 1] = 20;
    A[full_size - 1] = 30;
    A[grid_size * (grid_size - 1)] = 20;

    //заполняем(рассчитываем) рамку матрицы
    for (int i = 1; i < grid_size - 1; i++) {
        A[i] = 10 + i * step;
        A[i * (grid_size)] = 10 + i * step;
        A[grid_size * i + (grid_size - 1)] = 20 + i * step;
        A[grid_size * (grid_size - 1) + i] = 20 + i * step;
    }
    std::memcpy(Anew, A, sizeof(double) * full_size);
	double* error;
	hipHostMalloc(&error, sizeof(double));
	*error = 1.0;
	double min_error = accuracy;
    int max_iter = iters, iter = 0;

/////////////////////////////////////////////////////////////////////////////////
    double* ptr_A, *ptr_Anew, *deviceError, *errMx, *buff = NULL;
	size_t sizeofBuff = 0;    
    //выделяем память на gpu для будущих действий, проверяем статусы используемых функций (на всякий случай)
    //место в памяти для матриц
    hipError_t cudaStatus_1 = hipMalloc((void**)(&ptr_A), sizeof(double) * full_size);
	hipError_t cudaStatus = hipMalloc((void**)(&ptr_Anew), sizeof(double) * full_size);
	hipError_t cudaStatus_2 = hipMalloc((void**)&errMx, sizeof(double) * full_size);
	if (cudaStatus_1 != 0 || cudaStatus != 0 || cudaStatus_2 != 0){
		std::cout << "Pu-pu-pu, something is wrong with memory allocation" << std::endl;
		if (ptr_A != NULL) hipFree(ptr_A);
		if (ptr_Anew != NULL) hipFree(ptr_Anew);
		if (errMx != NULL) hipFree(errMx);
		return 42;

		//освободить память ++++++++++++++++++++++++++++++++++++++++++++++++++++
	}    
    //место в памяти для переменной ошибки
    cudaStatus_2 = hipMalloc((void**)&deviceError, sizeof(double));

	cudaStatus_1 = hipMemcpy(ptr_A, A, sizeof(double) * full_size, hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(ptr_Anew, Anew, sizeof(double) * full_size, hipMemcpyHostToDevice);
	if (cudaStatus_1 != 0 || cudaStatus != 0 || cudaStatus_2 != 0){
		if (ptr_A != NULL) hipFree(ptr_A);
		if (ptr_Anew != NULL) hipFree(ptr_Anew);
		if (errMx != NULL) hipFree(errMx);
		if (deviceError != NULL) hipFree(deviceError);
		std::cout << "Pu-pu-pu, something is wrong with memory transfer" << std::endl;
		return 42;
	}//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! (IF)

	//получаем значение sizeofBuff, чтобы выделить память для buff 
    // buff пригодится для промежуточных вычислений в функции Max
    // на данном этапе Max ничего не меняет, кроме sizeofBuff, так как *buff = NULL
	hipcub::DeviceReduce::Max(buff, sizeofBuff, errMx, deviceError, full_size);
	hipMalloc((void**)&buff, sizeofBuff);



    bool graphFlag = false;
	hipStream_t stream;
	hipStreamCreate(&stream);
	hipGraph_t graph;
	hipGraphExec_t instance;
    nvtxRangePushA("pepe");



	//////
	size_t threads;
	if (grid_size < 1024) threads= grid_size;
	else threads = 1024;
    unsigned int blocks = grid_size / threads;
	dim3 blockD(threads / 32, threads / 32);
    dim3 gridD(blocks * 32, blocks * 32);
	//////
	
		while(iter < max_iter && *error > min_error)
	{
    //основные вычисление с использованием нескольких ядер
    //кол-во потоков: (grid_size - 1)^2
    	if (graphFlag){
			//запускаем выполнения заданных нод
			hipGraphLaunch(instance, stream);
			//hipStreamSynchronize(stream);
			hipMemcpyAsync(error, deviceError, sizeof(double), hipMemcpyDeviceToHost, stream);
			//ждем, пока выполнится граф
			hipStreamSynchronize(stream);
			iter += 100;
		}
		else{
			//захватываем граф, наинаем добавлять ноды (функции)
			hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

			for(size_t i = 0; i < 50; i++)
			{
				calculateMatrix<<<gridD, blockD, 0, stream>>>(ptr_A, ptr_Anew, grid_size);
				calculateMatrix<<<gridD, blockD, 0, stream>>>(ptr_Anew, ptr_A, grid_size);
			}
			// Расчитываем ошибку каждую сотую итерацию
			getErrorMatrix<<<threads*blocks*blocks, threads,  0, stream>>>(ptr_A, ptr_Anew, errMx, grid_size);
			hipcub::DeviceReduce::Max(buff, sizeofBuff, errMx, deviceError, full_size, stream);
			//заканчиваем работать с графом
			hipStreamEndCapture(stream, &graph);

			//после закидывания нод, инициализируем (создаем) сам граф
			hipGraphInstantiate(&instance, graph, NULL, NULL, 0);
			graphFlag = true;
  		}
}
	
	
    nvtxRangePop();
	std::cout << "Iter: " << iter << std::endl;
    std::cout << "Error: " << *error << std::endl;

    //обязательно освобождаем память
    free(A);    
    free(Anew);
	hipFree(ptr_A);
	hipFree(ptr_Anew);
	hipStreamDestroy(stream);
	hipGraphDestroy(graph);
	hipFree(errMx);
	hipFree(buff);
    return 0;
}